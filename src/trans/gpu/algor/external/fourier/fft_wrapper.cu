// (C) Copyright 2022- NVIDIA.
//
// This software is licensed under the terms of the Apache Licence Version 2.0
// which can be obtained at http://www.apache.org/licenses/LICENSE-2.0.
// In applying this licence, ECMWF does not waive the privileges and immunities
// granted to it by virtue of its status as an intergovernmental organisation
// nor does it submit to any jurisdiction.


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
#include <iostream>
#include <unordered_map>
#include <vector>

static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}
#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

#define CUFFT_CHECK(e)                                                         \
  {                                                                            \
    hipfftResult_t err = (e);                                                   \
    if (err != HIPFFT_SUCCESS) {                                                \
      fprintf(stderr, "CUFFT error: %s, line %d, %s: %s\n", __FILE__,          \
              __LINE__, #e, _cudaGetErrorEnum(err));                           \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

extern void *planWorkspace;

namespace {
struct Double {
  using real = double;
  using cmplx = hipfftDoubleComplex;
};
struct Float {
  using real = float;
  using cmplx = hipfftComplex;
};
} // namespace
template <class Type, hipfftType Direction>
void execute_fft(typename Type::real *data_real,
                 typename Type::cmplx *data_complex, int kfield, int *loens,
                 int *offsets, int nfft) {
  constexpr bool is_forward = Direction == HIPFFT_R2C || Direction == HIPFFT_D2Z;
  using real = typename Type::real;
  using cmplx = typename Type::cmplx;

  /* static std::unordered_map<int, void *> allocationCache; // nloens -> ptr */
  static std::unordered_map<int, std::vector<hipfftHandle>>
      fftPlansCache; // kfield -> handles

  // if the pointers are changed, we need to update the graph
  static std::unordered_map<int, std::pair<real *, cmplx *>>
      ptrCache; // kfield -> ptrs

  auto ptrs = ptrCache.find(kfield);
  if (ptrs != ptrCache.end() && (ptrs->second.first != data_real ||
                                 ptrs->second.second != data_complex)) {
    // the plan is cached, but the pointers are not correct. we remove and
    // delete the graph, but we keep the FFT plans, if this happens more often,
    // we should cache this...
    std::cout << "WARNING FFT: POINTER CHANGE --> THIS MIGHT BE SLOW" << std::endl;
    ptrCache.erase(kfield);
  }

    auto fftPlans = fftPlansCache.find(kfield);
    if (fftPlans == fftPlansCache.end()) {
      // the fft plans do not exist yet
      std::vector<hipfftHandle> newPlans;
      newPlans.resize(nfft);
      for (int i = 0; i < nfft; ++i) {
        int nloen = loens[i];

        hipfftHandle plan;
        CUFFT_CHECK(hipfftCreate(&plan));
        int dist = offsets[i + 1] - offsets[i];
        int embed[] = {1};
        CUFFT_CHECK(hipfftPlanMany(
            &plan, 1, &nloen, embed, 1, is_forward ? dist : dist / 2, embed, 1,
            is_forward ? dist / 2 : dist, Direction, kfield));
        newPlans[i] = plan;
      }
      fftPlansCache.insert({kfield, newPlans});
    }
    fftPlans = fftPlansCache.find(kfield);

    for (int i = 0; i < nfft; ++i) {
      int offset = offsets[i];
      real *data_real_l = &data_real[kfield * offset];
      cmplx *data_complex_l = &data_complex[kfield * offset / 2];
      if constexpr (Direction == HIPFFT_R2C)
        CUFFT_CHECK(
            hipfftExecR2C(fftPlans->second[i], data_real_l, data_complex_l))
      else if constexpr (Direction == HIPFFT_C2R)
        CUFFT_CHECK(
            hipfftExecC2R(fftPlans->second[i], data_complex_l, data_real_l))
      else if constexpr (Direction == HIPFFT_D2Z)
        CUFFT_CHECK(
            hipfftExecD2Z(fftPlans->second[i], data_real_l, data_complex_l))
      else if constexpr (Direction == HIPFFT_Z2D)
        CUFFT_CHECK(
            hipfftExecZ2D(fftPlans->second[i], data_complex_l, data_real_l));
    }

    ptrCache.insert({kfield, std::make_pair(data_real, data_complex)});
  CUDA_CHECK(hipDeviceSynchronize());
}
extern "C" {
void execute_dir_fft_float(float *data_real, hipfftComplex *data_complex,
                           int kfield, int *loens, int *offsets, int nfft) {
  execute_fft<Float, HIPFFT_R2C>(data_real, data_complex, kfield, loens, offsets,
                                nfft);
}
void execute_inv_fft_float(hipfftComplex *data_complex, float *data_real,
                           int kfield, int *loens, int *offsets, int nfft) {
  execute_fft<Float, HIPFFT_C2R>(data_real, data_complex, kfield, loens, offsets,
                                nfft);
}
void execute_dir_fft_double(double *data_real, hipfftDoubleComplex *data_complex,
                            int kfield, int *loens, int *offsets, int nfft) {
  execute_fft<Double, HIPFFT_D2Z>(data_real, data_complex, kfield, loens,
                                 offsets, nfft);
}
void execute_inv_fft_double(hipfftDoubleComplex *data_complex, double *data_real,
                            int kfield, int *loens, int *offsets, int nfft) {
  execute_fft<Double, HIPFFT_Z2D>(data_real, data_complex, kfield, loens,
                                 offsets, nfft);
}
}
